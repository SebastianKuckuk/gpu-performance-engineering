#include "hip/hip_runtime.h"
#include "stream-util.h"

#include "../cuda-util.h"


template <typename tpe>
__global__ void stream(const tpe *const __restrict__ src, tpe *__restrict__ dest, size_t nx) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x;

    if (i0 < nx) {
        dest[i0] = src[i0] + 1;
    }
}


template <typename tpe>
inline int realMain(int argc, char *argv[]) {
    char *tpeName;
    size_t nx, nItWarmUp, nIt;
    parseCLA_1d(argc, argv, tpeName, nx, nItWarmUp, nIt);

    tpe *dest;
    checkCudaError(hipHostMalloc((void **)&dest, sizeof(tpe) * nx));
    tpe *src;
    checkCudaError(hipHostMalloc((void **)&src, sizeof(tpe) * nx));

    tpe *d_dest;
    checkCudaError(hipMalloc((void **)&d_dest, sizeof(tpe) * nx));
    tpe *d_src;
    checkCudaError(hipMalloc((void **)&d_src, sizeof(tpe) * nx));

    // init
    initStream(dest, src, nx);

    checkCudaError(hipMemcpy(d_dest, dest, sizeof(tpe) * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_src, src, sizeof(tpe) * nx, hipMemcpyHostToDevice));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        stream<<<ceilingDivide(nx, 256), 256>>>(d_src, d_dest, nx);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        stream<<<ceilingDivide(nx, 256), 256>>>(d_src, d_dest, nx);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats<tpe>(end - start, nIt, nx, tpeName, sizeof(tpe) + sizeof(tpe), 1);

    checkCudaError(hipMemcpy(dest, d_dest, sizeof(tpe) * nx, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(src, d_src, sizeof(tpe) * nx, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionStream(dest, src, nx, nIt + nItWarmUp);

    checkCudaError(hipFree(d_dest));
    checkCudaError(hipFree(d_src));

    checkCudaError(hipHostFree(dest));
    checkCudaError(hipHostFree(src));

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("int" == tpeName)
        return realMain<int>(argc, argv);
    if ("long" == tpeName)
        return realMain<long>(argc, argv);
    if ("float" == tpeName)
        return realMain<float>(argc, argv);
    if ("double" == tpeName)
        return realMain<double>(argc, argv);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  int, long, float, double" << std::endl;
    return -1;
}
