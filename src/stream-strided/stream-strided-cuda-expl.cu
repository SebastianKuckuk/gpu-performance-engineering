#include "hip/hip_runtime.h"
#include "stream-strided-util.h"

#include "../cuda-util.h"


template <typename tpe>
__global__ void streamstrided(const tpe *const __restrict__ src, tpe *__restrict__ dest, size_t nx, size_t strideRead, size_t strideWrite) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x;

    if (i0 < nx) {
        dest[i0 * strideWrite] = src[i0 * strideRead] + 1;
    }
}


template <typename tpe>
inline int realMain(int argc, char *argv[]) {
    char *tpeName;
    size_t nx, nItWarmUp, nIt;
    size_t strideRead;
    size_t strideWrite;
    parseCLA_1d(argc, argv, tpeName, nx, strideRead, strideWrite, nItWarmUp, nIt);

    tpe *dest;
    checkCudaError(hipHostMalloc((void **)&dest, sizeof(tpe) * nx * std::max(strideRead, strideWrite)));
    tpe *src;
    checkCudaError(hipHostMalloc((void **)&src, sizeof(tpe) * nx * std::max(strideRead, strideWrite)));

    tpe *d_dest;
    checkCudaError(hipMalloc((void **)&d_dest, sizeof(tpe) * nx * std::max(strideRead, strideWrite)));
    tpe *d_src;
    checkCudaError(hipMalloc((void **)&d_src, sizeof(tpe) * nx * std::max(strideRead, strideWrite)));

    // init
    initStreamStrided(dest, src, nx, strideRead, strideWrite);

    checkCudaError(hipMemcpy(d_dest, dest, sizeof(tpe) * nx * std::max(strideRead, strideWrite), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_src, src, sizeof(tpe) * nx * std::max(strideRead, strideWrite), hipMemcpyHostToDevice));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        streamstrided<<<ceilingDivide(nx, 256), 256>>>(d_src, d_dest, nx, strideRead, strideWrite);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        streamstrided<<<ceilingDivide(nx, 256), 256>>>(d_src, d_dest, nx, strideRead, strideWrite);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats<tpe>(end - start, nIt, nx, tpeName, sizeof(tpe) + sizeof(tpe), 1);

    checkCudaError(hipMemcpy(dest, d_dest, sizeof(tpe) * nx * std::max(strideRead, strideWrite), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(src, d_src, sizeof(tpe) * nx * std::max(strideRead, strideWrite), hipMemcpyDeviceToHost));

    // check solution
    checkSolutionStreamStrided(dest, src, nx, nIt + nItWarmUp, strideRead, strideWrite);

    checkCudaError(hipFree(d_dest));
    checkCudaError(hipFree(d_src));

    checkCudaError(hipHostFree(dest));
    checkCudaError(hipHostFree(src));

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("int" == tpeName)
        return realMain<int>(argc, argv);
    if ("long" == tpeName)
        return realMain<long>(argc, argv);
    if ("float" == tpeName)
        return realMain<float>(argc, argv);
    if ("double" == tpeName)
        return realMain<double>(argc, argv);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  int, long, float, double" << std::endl;
    return -1;
}
