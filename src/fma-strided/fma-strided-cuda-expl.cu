#include "hip/hip_runtime.h"
#include "fma-strided-util.h"

#include "../cuda-util.h"


template <typename tpe>
__global__ void fmastrided(tpe *__restrict__ data, size_t nx, size_t stride) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x;

    if (i0 < nx * stride) {
        tpe a = (tpe)0.5, b = (tpe)1;
        // dummy op to prevent compiler from solving loop analytically
        if (1 == nx) {
            auto tmp = b;
            b = a;
            a = tmp;
        }

        tpe acc = i0;

        if (0 == i0 % stride) {
            acc = data[i0];
            for (auto r = 0; r < 65536; ++r)
                acc = a * acc + b;
        }

        // dummy check to prevent compiler from eliminating loop
        if ((tpe)0 == acc)
            data[i0 / stride] = acc;
    }
}


template <typename tpe>
inline int realMain(int argc, char *argv[]) {
    char *tpeName;
    size_t nx, nItWarmUp, nIt;
    size_t stride;
    parseCLA_1d(argc, argv, tpeName, nx, stride, nItWarmUp, nIt);

    tpe *data;
    checkCudaError(hipHostMalloc((void **)&data, sizeof(tpe) * nx * stride));

    tpe *d_data;
    checkCudaError(hipMalloc((void **)&d_data, sizeof(tpe) * nx * stride));

    // init
    initFmaStrided(data, nx, stride);

    checkCudaError(hipMemcpy(d_data, data, sizeof(tpe) * nx * stride, hipMemcpyHostToDevice));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        fmastrided<<<ceilingDivide(nx * stride, 256), 256>>>(d_data, nx, stride);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        fmastrided<<<ceilingDivide(nx * stride, 256), 256>>>(d_data, nx, stride);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats<tpe>(end - start, nIt, nx, tpeName, sizeof(tpe), 131072);

    checkCudaError(hipMemcpy(data, d_data, sizeof(tpe) * nx * stride, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionFmaStrided(data, nx, nIt + nItWarmUp, stride);

    checkCudaError(hipFree(d_data));

    checkCudaError(hipHostFree(data));

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("int" == tpeName)
        return realMain<int>(argc, argv);
    if ("long" == tpeName)
        return realMain<long>(argc, argv);
    if ("float" == tpeName)
        return realMain<float>(argc, argv);
    if ("double" == tpeName)
        return realMain<double>(argc, argv);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  int, long, float, double" << std::endl;
    return -1;
}
